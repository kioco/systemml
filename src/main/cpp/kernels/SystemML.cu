/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/**********************************
When updating a kernel or adding a new one,
please compile the ptx file and commit it:
nvcc -ptx -arch=sm_30 --std c++11 SystemML.cu
***********************************/


#include <hip/hip_runtime.h>
#include <cfloat>
#include <cmath>

extern "C" __global__ void double2float_f(double *A, float *ret, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < N) {
    // TODO: Use __double2float_rd or __double2float_rn  or __double2float_ru or
    // __double2float_rz after
    ret[tid] = (float)A[tid];
  }
}

extern "C" __global__ void float2double_f(float *A, double *ret, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < N) {
    ret[tid] = (double)A[tid];
  }
}

/**
 * Performs a slice operation where the input matrix is sparse and the output
 * matrix is dense.
 * This function avoids unnecessary sparse to dense conversion of the input
 * matrix.
 * Parallelization: rows of output matrix.
 *
 * @params inVal input val pointer
 * @params inRowPtr input row pointer
 * @params colInd input col index pointer
 * @params ret dense output pointer
 * @param rl row lower
 * @param ru row upper
 * @param cl column lower
 * @param cu column upper
 * @param retClen number of columns of output matrix
 */
template <typename T>
__device__ void slice_sparse_dense_row(T *inVal, int *inRowPtr, int *colInd,
                                       T *ret, int rl, int ru, int cl, int cu,
                                       int retClen) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int rowIndex = index + rl;
  if (rowIndex <= ru) {
    /*
     * TODO: Alternative approach: use dynamic parallelism. We are skipping this
*for now to avoid
     * the complexity of two-step separate compilation and linking process.
     *
     * extern "C"
     * __global__ void slice_sparse_dense_row_helper(double* inVal, int*
*inRowPtr, int* colInd, double* ret,
     *     int rl, int ru, int cl, int cu, int retClen, int start, int end, int
*index) {
     *  int i = blockIdx.x * blockDim.x + threadIdx.x + start;
     * 	// Only slice if the index falls into the given range
     * 	if(i < end && cl <= colInd[i] && colInd[i] <= cu) {
     * 		ret[ index*retClen + (colInd[i] - cl) ] = inVal[i];
     * 	}
     * }
     *
     * int size = inRowPtr[rowIndex+1] - inRowPtr[rowIndex];
     * double numThreads = (double)min(size, MAX_NUM_THREADS_CHILD_KERNEL);
     * slice_sparse_dense_row_helper
     * <<< ceil(numThreads/MAX_NUM_THREADS_CHILD_KERNEL), MAX_NUM_THREADS_CHILD_KERNEL>>>
     * (inVal, inRowPtr, colInd, ret, rl, ru, cl, cu, retClen, inRowPtr[rowIndex],
     *	inRowPtr[rowIndex+1], index);
     *
     * Two-step compilation and linking process in JCudaKernels's constructor:
     * cuLinkAddFile(linkState, CUjitInputType.CU_JIT_INPUT_LIBRARY,
     * "/usr/local/cuda/lib64/libcudadevrt.a", jitOptions);
     */
    // Iterate over elements of the row 'rowIndex'.
    for (int i = inRowPtr[rowIndex]; i < inRowPtr[rowIndex + 1]; i++) {
      // Only slice if the index falls into the given range
      if (cl <= colInd[i] && colInd[i] <= cu) {
        ret[index * retClen + (colInd[i] - cl)] = inVal[i];
      }
    }
  }
}

extern "C" __global__ void slice_sparse_dense_row_d(double *inVal,
                                                    int *inRowPtr, int *colInd,
                                                    double *ret, int rl, int ru,
                                                    int cl, int cu,
                                                    int retClen) {
  slice_sparse_dense_row(inVal, inRowPtr, colInd, ret, rl, ru, cl, cu, retClen);
}

extern "C" __global__ void slice_sparse_dense_row_f(float *inVal, int *inRowPtr,
                                                    int *colInd, float *ret,
                                                    int rl, int ru, int cl,
                                                    int cu, int retClen) {
  slice_sparse_dense_row(inVal, inRowPtr, colInd, ret, rl, ru, cl, cu, retClen);
}

/**
 * Performs a slice operation where the input matrix is sparse and the output
 * matrix is dense.
 * This function avoids unnecessary sparse to dense conversion of the input
 * matrix.
 * Parallelization: subset of number of non-zeroes of input matrix.
 *
 * @params inVal input val pointer
 * @params inRowPtr input row pointer
 * @params colInd input col index pointer
 * @params ret dense output pointer
 * @param rl row lower
 * @param ru row upper
 * @param cl column lower
 * @param cu column upper
 * @param retClen number of columns of output matrix
 */
template <typename T>
__device__ void slice_sparse_dense_nnz(T *inVal, int *inRowPtr, int *colInd,
                                       T *ret, int rl, int ru, int cl, int cu,
                                       int retClen) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int i = tid + inRowPtr[rl];

  // Only slice if the index falls into the given range
  if (i < inRowPtr[ru + 1] && cl <= colInd[i] && colInd[i] <= cu) {
    // Find the row index for corresponding non-zero value 'i'.
    int rowIndex = rl;
    while (inRowPtr[rowIndex + 1] <= i) {
      rowIndex++;
    }
    ret[(rowIndex - rl) * retClen + (colInd[i] - cl)] = inVal[i];
  }
}

extern "C" __global__ void slice_sparse_dense_nnz_d(double *inVal,
                                                    int *inRowPtr, int *colInd,
                                                    double *ret, int rl, int ru,
                                                    int cl, int cu,
                                                    int retClen) {
  slice_sparse_dense_nnz(inVal, inRowPtr, colInd, ret, rl, ru, cl, cu, retClen);
}

extern "C" __global__ void slice_sparse_dense_nnz_f(float *inVal, int *inRowPtr,
                                                    int *colInd, float *ret,
                                                    int rl, int ru, int cl,
                                                    int cu, int retClen) {
  slice_sparse_dense_nnz(inVal, inRowPtr, colInd, ret, rl, ru, cl, cu, retClen);
}

/**
 * Performs a slice operation where the input matrix is dense and the output
 * matrix is dense.
 *
 * @params in dense input pointer
 * @params ret dense output pointer
 * @param rl row lower
 * @param ru row upper
 * @param cl column lower
 * @param cu column upper
 * @param inClen number of columns of input matrix
 * @param retRlen number of rows of output matrix
 * @param retClen number of columns of output matrix
 */
template <typename T>
__device__ void slice_dense_dense(T *in, T *ret, int rl, int ru, int cl, int cu,
                                  int inClen, int retRlen, int retClen) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int ix = tid / retClen;
  int iy = tid % retClen;
  if (ix < retRlen && iy < retClen) {
    int inIndex = (ix + rl) * inClen + cl + iy;
    ret[tid] = in[inIndex];
  }
}

extern "C" __global__ void slice_dense_dense_d(double *in, double *ret, int rl,
                                               int ru, int cl, int cu,
                                               int inClen, int retRlen,
                                               int retClen) {
  slice_dense_dense(in, ret, rl, ru, cl, cu, inClen, retRlen, retClen);
}

extern "C" __global__ void slice_dense_dense_f(float *in, float *ret, int rl,
                                               int ru, int cl, int cu,
                                               int inClen, int retRlen,
                                               int retClen) {
  slice_dense_dense(in, ret, rl, ru, cl, cu, inClen, retRlen, retClen);
}

/**
 * Does a copy of upper to lower triangle of the given matrix
 * @param ret the input and output array allocated on the GPU
 * @param dim the number of rows of the square matrix ret
 * @param N total number of elements of the matrix
 */
template <typename T>
__device__ void copy_u2l_dense(T *ret, int dim, int N) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int ix = tid / dim;
  int iy = tid % dim;
  int id_dest = iy * dim + ix;
  if (iy > ix && id_dest < N) {
    // TODO: Potential to reduce the number of threads by half
    int id_src = tid;
    ret[id_dest] = ret[id_src];
  }
}

extern "C" __global__ void copy_u2l_dense_d(double *ret, int dim, int N) {
  copy_u2l_dense(ret, dim, N);
}

extern "C" __global__ void copy_u2l_dense_f(float *ret, int dim, int N) {
  copy_u2l_dense(ret, dim, N);
}

// Use this method in templates to fetch the maximum value for a given datatype
template <typename T>
__forceinline__ __device__ T MAX() {
  return T();
}
template <>
__forceinline__ __device__ float MAX <float>() {
  return FLT_MAX;
}
template <>
__forceinline__ __device__ double MAX <double>() {
  return DBL_MAX;
}

// op = {0=plus, 1=minus, 2=multiply, 3=divide, 4=power,
// 5=less, 6=lessequal, 7=greater, 8=greaterequal, 9=equal, 10=notequal,
// 11=min, 12=max, 13=and, 14=or, 15=minus1multiply, 16=minusnz,
// 17=modulus, 18=integer division}
template <typename T>
__forceinline__ __device__ T binaryOp(T x, T y, int op) {
  switch (op) {
    case 0:
      return x + y;
    case 1:
      return x - y;
    case 2:
      return x * y;
    case 3:
      return x / y;
    case 4:
      return pow(x, y);
    case 5:
      return (x < y) == 0 ? 0.0 : 1.0;
    case 6:
      return (x <= y) == 0 ? 0.0 : 1.0;
    case 7:
      return (x > y) == 0 ? 0.0 : 1.0;
    case 8:
      return (x >= y) == 0 ? 0.0 : 1.0;
    case 9:
      return (x == y) == 0 ? 0.0 : 1.0;
    case 10:
      return (x != y) == 0 ? 0.0 : 1.0;
    case 11:
      return min(x, y);
    case 12:
      return max(x, y);
    case 13:
      return ((int)llrint(x) & (int)llrint(y)) == 0 ? 0.0 : 1.0;
    case 14:
      return ((int)llrint(x) | (int)llrint(y)) == 0 ? 0.0 : 1.0;
    case 15:
      return 1 - x * y;
    case 16:
      return (x != 0.0 ? x - y : 0.0);
    case 17: {
      if (y == 0.0 || y == -0.0) {
        return nan("");
      }
      T v = x / y;
      // Check for v being NaN (v != v) or if it is infinity
      if (isnan(v) || isinf(v)) {
        return v;
      } else {
        v = floor(v);
      }
      return x - v * y;
    }
    case 18: {
      T v = x / y;
      if (isnan(v) || isinf(v)) {
        return v;
      } else {
        return floor(v);
      }
    }
    default:
      return MAX<T>();
  }
}

/**
 * Performs forward pass for relu: ret = max(A, 0)
 *
 * @param A input array allocated on the GPU
 * @param ret output array allocated on the GPU
 * @param rlen the number of rows
 * @param clen the number of columns
 */
template <typename T>
__device__ void relu(T *A, T *ret, int rlen, int clen) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int ix = tid / clen;
  int iy = tid % clen;
  if (ix < rlen && iy < clen) {
    ret[tid] = max(0.0, A[tid]);
  }
}

extern "C" __global__ void relu_d(double *A, double *ret, int rlen, int clen) {
  relu(A, ret, rlen, clen);
}

extern "C" __global__ void relu_f(float *A, float *ret, int rlen, int clen) {
  relu(A, ret, rlen, clen);
}

/**
 * This method computes the backpropagation errors for previous layer of relu
 * operation
 *
 * @param X input activation array allocated on the GPU
 * @param dout errors from previous layer
 * @param ret output array allocated on the GPU
 * @param rlen the number of rows
 * @param clen the number of columns
 */
template <typename T>
__device__ void relu_backward(T *X, T *dout, T *ret, int rlen, int clen) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int ix = tid / clen;
  int iy = tid % clen;
  if (ix < rlen && iy < clen) {
    ret[tid] = X[tid] > 0 ? dout[tid] : 0;
  }
}

extern "C" __global__ void relu_backward_d(double *X, double *dout, double *ret,
                                           int rlen, int clen) {
  relu_backward(X, dout, ret, rlen, clen);
}

extern "C" __global__ void relu_backward_f(float *X, float *dout, float *ret,
                                           int rlen, int clen) {
  relu_backward(X, dout, ret, rlen, clen);
}

/**
 * Performs inplace addition: ret += input
 *
 * @param input rhs input array allocated on the GPU
 * @param ret the input and output array allocated on the GPU
 * @param rlen the number of rows
 * @param clen the number of columns
 */
template <typename T>
__device__ void inplace_add(T *input, T *ret, int rlen, int clen) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int ix = tid / clen;
  int iy = tid % clen;
  if (ix < rlen && iy < clen) {
    ret[tid] += input[tid];
  }
}

extern "C" __global__ void inplace_add_d(double *input, double *ret, int rlen,
                                         int clen) {
  inplace_add(input, ret, rlen, clen);
}

extern "C" __global__ void inplace_add_f(float *input, float *ret, int rlen,
                                         int clen) {
  inplace_add(input, ret, rlen, clen);
}

// Performs the operation corresponding to the DML script:
// ones = matrix(1, rows=1, cols=Hout*Wout)
// output = input + matrix(bias %*% ones, rows=1, cols=F*Hout*Wout)
// This operation is often followed by conv2d and hence we have introduced
// bias_add(input, bias) built-in function
template <typename T>
__device__ void bias_add(T *input, T *bias, T *ret, int rlen, int clen,
                         int PQ) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int ix = tid / clen;
  int iy = tid % clen;
  if (ix < rlen && iy < clen) {
    int biasIndex = iy / PQ;
    ret[tid] = input[tid] + bias[biasIndex];
  }
}

extern "C" __global__ void bias_add_d(double *input, double *bias, double *ret,
                                      int rlen, int clen, int PQ) {
  bias_add(input, bias, ret, rlen, clen, PQ);
}

extern "C" __global__ void bias_add_f(float *input, float *bias, float *ret,
                                      int rlen, int clen, int PQ) {
  bias_add(input, bias, ret, rlen, clen, PQ);
}

// Performs the operation "ret <- A + alpha*B", where B is a vector
template <typename T>
__device__ void daxpy_matrix_vector(T *A, T *B, double alpha, T *ret, int rlenA,
                                    int clenA, int rlenB, int clenB) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int ix = tid / clenA;
  int iy = tid % clenA;
  if (ix < rlenA && iy < clenA) {
    int index = ix * clenA + iy;
    if (rlenB == 1) {
      ret[index] = A[index] + alpha * B[iy];
    } else {
      ret[index] = A[index] + alpha * B[ix];
    }
  }
}

extern "C" __global__ void daxpy_matrix_vector_d(double *A, double *B,
                                                 double alpha, double *ret,
                                                 int rlenA, int clenA,
                                                 int rlenB, int clenB) {
  daxpy_matrix_vector(A, B, alpha, ret, rlenA, clenA, rlenB, clenB);
}

extern "C" __global__ void daxpy_matrix_vector_f(float *A, float *B,
                                                 double alpha, float *ret,
                                                 int rlenA, int clenA,
                                                 int rlenB, int clenB) {
  daxpy_matrix_vector(A, B, alpha, ret, rlenA, clenA, rlenB, clenB);
}

// Performs similar operation as bias_add except elementwise multiplication
// instead of add
template <typename T>
__device__ void bias_multiply(T *input, T *bias, T *ret, int rlen, int clen,
                              int PQ) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int ix = tid / clen;
  int iy = tid % clen;
  if (ix < rlen && iy < clen) {
    int biasIndex = iy / PQ;
    ret[tid] = input[tid] * bias[biasIndex];
  }
}

extern "C" __global__ void bias_multiply_d(double *input, double *bias,
                                           double *ret, int rlen, int clen,
                                           int PQ) {
  bias_multiply(input, bias, ret, rlen, clen, PQ);
}

extern "C" __global__ void bias_multiply_f(float *input, float *bias,
                                           float *ret, int rlen, int clen,
                                           int PQ) {
  bias_multiply(input, bias, ret, rlen, clen, PQ);
}

/**
 * Performs a binary cellwise arithmetic operation on 2 matrices.
 * Either both matrices are of equal size or one of them is a vector or both
 * are.
 * @param A                 first input matrix allocated on GPU
 * @param B                 second input matrix allocated on GPU
 * @param C                 output allocated on GPU
 * @param maxRlen           maximum of the row lengths of A and B
 * @param maxClen           maximum of the column lengths of A and B
 * @param vectorAStatus     if A is a row vector, column vector or neither
 * @param vectorBStatus     if B is a row vector, column vector or neither
 * @param op                the numeric code of the arithmetic operation to
 * perform
 *
 */
template <typename T>
__device__ void matrix_matrix_cellwise_op(T *A, T *B, T *C, int maxRlen,
                                          int maxClen, int vectorAStatus,
                                          int vectorBStatus, int op) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int ix = tid / maxClen;
  int iy = tid % maxClen;

  if (ix < maxRlen && iy < maxClen) {
    int outIndex = ix * maxClen + iy;
    int aIndex = outIndex;
    int bIndex = outIndex;
    if (vectorAStatus == 1)
      aIndex = ix;  // clen == 1
    else if (vectorAStatus == 2)
      aIndex = iy;  // rlen == 1
    if (vectorBStatus == 1)
      bIndex = ix;  // clen == 1
    else if (vectorBStatus == 2)
      bIndex = iy;  // rlen == 1
    C[outIndex] = binaryOp(A[aIndex], B[bIndex], op);
    // printf("C[%d] = A[%d](%f) B[%d](%f) (%d %d)\n", outIndex, aIndex,
    // A[aIndex], bIndex,  B[bIndex], (ix+1), (iy+1));
    __syncthreads();
  }
}

extern "C" __global__ void matrix_matrix_cellwise_op_d(
    double *A, double *B, double *C, int maxRlen, int maxClen,
    int vectorAStatus, int vectorBStatus, int op) {
  matrix_matrix_cellwise_op(A, B, C, maxRlen, maxClen, vectorAStatus,
                            vectorBStatus, op);
}

extern "C" __global__ void matrix_matrix_cellwise_op_f(
    float *A, float *B, float *C, int maxRlen, int maxClen, int vectorAStatus,
    int vectorBStatus, int op) {
  matrix_matrix_cellwise_op(A, B, C, maxRlen, maxClen, vectorAStatus,
                            vectorBStatus, op);
}

/**
 * Performs an arithmetic operation between a matrix and a scalar.
 * C = s op A or C = A op s (where A is the matrix, s is the scalar and op is
 * the operation)
 * @param A             input matrix allocated on GPU
 * @param scalar        scalar input
 * @param C             output matrix allocated on GPU
 * @param size          number of elements in matrix A
 * @param op            number code of the arithmetic operation to perform
 * @param isLeftScalar  whether the scalar is on the left side
 */
template <typename T>
__device__ void matrix_scalar_op(T *A, T scalar, T *C, int size, int op,
                                 int isLeftScalar) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    if (isLeftScalar) {
      C[index] = binaryOp(scalar, A[index], op);
    } else {
      C[index] = binaryOp(A[index], scalar, op);
    }
  }
  __syncthreads();
}

extern "C" __global__ void matrix_scalar_op_d(double *A, double scalar,
                                              double *C, int size, int op,
                                              int isLeftScalar) {
  matrix_scalar_op(A, scalar, C, size, op, isLeftScalar);
}

extern "C" __global__ void matrix_scalar_op_f(float *A, double scalar, float *C,
                                              int size, int op,
                                              int isLeftScalar) {
  matrix_scalar_op(A, (float)scalar, C, size, op, isLeftScalar);
}

/**
 * Sets all elements (fills) of a double array of given length with a given
 * scalar value
 * @param A         array to be filled
 * @param scalar    value to fill array with
 * @param lenA      length of array A
 */
template <typename T>
__device__ void fill(T *A, T scalar, int lenA) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < lenA) {
    A[index] = scalar;
  }
}

extern "C" __global__ void fill_d(double *A, double scalar, int lenA) {
  fill(A, scalar, lenA);
}

extern "C" __global__ void fill_f(float *A, double scalar, int lenA) {
  fill(A, (float)scalar, lenA);
}

/**
 * Appends Matrix B to the right side of Matrix A into a new matrix C
 *         | 1 2 3 4 |   | 8 8 8 |     | 1 2 3 4 8 8 8 |
 * cbind ( | 9 8 7 6 | , | 7 7 7 | ) = | 9 8 7 6 7 7 7 |
 *         | 4 3 2 1 |   | 9 9 9 |     | 4 3 2 1 9 9 9 |
 * @param A      input matrix A allocated on the GPU
 * @param B      input matrix B allocated on the GPU
 * @param C      input matrix C allocated on the GPU
 * @param rowsA  rows in A
 * @param colsA  columns in A
 * @param rowsB  rows in B
 * @param colsB  columns in B
 */
template <typename T>
__device__ void cbind(T *A, T *B, T *C, int rowsA, int colsA, int rowsB,
                      int colsB) {
  int maxClen = max(colsA, colsB);
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int ix = tid / maxClen;
  int iy = tid % maxClen;

  int colsC = colsA + colsB;
  int rowsC = rowsA;

  // Copy an element of A into C into the appropriate location
  if (ix < rowsA && iy < colsA) {
    T elemA = A[ix * colsA + iy];
    C[ix * colsC + iy] = elemA;
  }

  // Copy an element of B into C into the appropriate location
  if (ix < rowsB && iy < colsB) {
    T elemB = B[ix * colsB + iy];
    C[ix * colsC + (iy + colsA)] = elemB;
  }
}

extern "C" __global__ void cbind_d(double *A, double *B, double *C, int rowsA,
                                   int colsA, int rowsB, int colsB) {
  cbind(A, B, C, rowsA, colsA, rowsB, colsB);
}

extern "C" __global__ void cbind_f(float *A, float *B, float *C, int rowsA,
                                   int colsA, int rowsB, int colsB) {
  cbind(A, B, C, rowsA, colsA, rowsB, colsB);
}

/**
 * Appends Matrix B to the bottom of Matrix A into a new matrix C
 *         | 2 3 4 |   | 8 8 8 |     | 2 3 4 |
 * rbind ( | 8 7 6 | , | 7 7 7 | ) = | 8 7 6 |
 *         | 3 2 1 |                 | 3 2 1 |
                                     | 8 8 8 |
                                     | 7 7 7 |
 * @param A      input matrix A allocated on the GPU
 * @param B      input matrix B allocated on the GPU
 * @param C      input matrix C allocated on the GPU
 * @param rowsA  rows in A
 * @param colsA  columns in A
 * @param rowsB  rows in B
 * @param colsB  columns in B
 */
template <typename T>
__device__ void rbind(T *A, T *B, T *C, int rowsA, int colsA, int rowsB,
                      int colsB) {
  int maxClen = max(colsA, colsB);
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int ix = tid / maxClen;
  int iy = tid % maxClen;

  int rowsC = rowsA + rowsB;
  int colsC = colsA;

  // Copy an element of A into C into the appropriate location
  if (ix < rowsA && iy < colsA) {
    T elemA = A[ix * colsA + iy];
    C[ix * colsC + iy] = elemA;
  }

  // Copy an element of B into C into the appropriate location
  if (ix < rowsB && iy < colsB) {
    T elemB = B[ix * colsB + iy];
    C[(ix + rowsA) * colsC + iy] = elemB;
  }
}

extern "C" __global__ void rbind_d(double *A, double *B, double *C, int rowsA,
                                   int colsA, int rowsB, int colsB) {
  rbind(A, B, C, rowsA, colsA, rowsB, colsB);
}

extern "C" __global__ void rbind_f(float *A, float *B, float *C, int rowsA,
                                   int colsA, int rowsB, int colsB) {
  rbind(A, B, C, rowsA, colsA, rowsB, colsB);
}

/**
 * Does a reduce operation over all elements of the array.
 * This method has been adapted from the Reduction sample in the NVIDIA CUDA
 * Samples (v8.0)
 * and the Reduction example available through jcuda.org
 * When invoked initially, all blocks partly compute the reduction operation
 * over the entire array
 * and writes it to the output/temporary array. A second invokation needs to
 * happen to get the
 * reduced value.
 * The number of threads, blocks and amount of shared memory is calculated in a
 * specific way.
 * Please refer to the NVIDIA CUDA Sample or the SystemML code that invokes this
 * method to see
 * how its done.
 * The template-ized version of this function is similar to what is found in
 * NVIDIA CUB
 *
 * @param ReductionOp       Type of the functor object that implements the
 * reduction operation
 */
template <typename ReductionOp, typename T>
__device__ void reduce(
    T *g_idata,  ///< input data stored in device memory (of size n)
    T *g_odata,  ///< output/temporary array stored in device memory (of size n)
    unsigned int n,  ///< size of the input and temporary/output arrays
    ReductionOp
        reduction_op,  ///< Reduction operation to perform (functor object)
    T initialValue)    ///< initial value for the reduction variable
{
  // extern __shared__ T sdata[];
  extern __shared__ __align__(sizeof(T)) unsigned char my_sdata[];
  T *sdata = reinterpret_cast<T *>(my_sdata);

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
  unsigned int gridSize = blockDim.x * 2 * gridDim.x;

  T v = initialValue;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  while (i < n) {
    v = reduction_op(v, g_idata[i]);
    // ensure we don't read out of bounds
    if (i + blockDim.x < n) v = reduction_op(v, g_idata[i + blockDim.x]);
    i += gridSize;
  }

  // each thread puts its local sum into shared memory
  sdata[tid] = v;
  __syncthreads();

  // do reduction in shared mem
  if (blockDim.x >= 1024) {
    if (tid < 512) {
      sdata[tid] = v = reduction_op(v, sdata[tid + 512]);
    }
    __syncthreads();
  }
  if (blockDim.x >= 512) {
    if (tid < 256) {
      sdata[tid] = v = reduction_op(v, sdata[tid + 256]);
    }
    __syncthreads();
  }
  if (blockDim.x >= 256) {
    if (tid < 128) {
      sdata[tid] = v = reduction_op(v, sdata[tid + 128]);
    }
    __syncthreads();
  }
  if (blockDim.x >= 128) {
    if (tid < 64) {
      sdata[tid] = v = reduction_op(v, sdata[tid + 64]);
    }
    __syncthreads();
  }

  if (tid < 32) {
    // now that we are using warp-synchronous programming (below)
    // we need to declare our shared memory volatile so that the compiler
    // doesn't reorder stores to it and induce incorrect behavior.
    volatile T *smem = sdata;
    if (blockDim.x >= 64) {
      smem[tid] = v = reduction_op(v, smem[tid + 32]);
    }
    if (blockDim.x >= 32) {
      smem[tid] = v = reduction_op(v, smem[tid + 16]);
    }
    if (blockDim.x >= 16) {
      smem[tid] = v = reduction_op(v, smem[tid + 8]);
    }
    if (blockDim.x >= 8) {
      smem[tid] = v = reduction_op(v, smem[tid + 4]);
    }
    if (blockDim.x >= 4) {
      smem[tid] = v = reduction_op(v, smem[tid + 2]);
    }
    if (blockDim.x >= 2) {
      smem[tid] = v = reduction_op(v, smem[tid + 1]);
    }
  }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/**
 * Does a reduce (sum) over each row of the array.
 * This kernel must be launched with as many blocks as there are rows.
 * The intuition for this kernel is that each block does a reduction over a
 * single row.
 * The maximum number of blocks that can launched (as of compute capability 3.0)
 * is 2^31 - 1
 * This works out fine for SystemML, since the maximum elements in a Java array
 * can be 2^31 - c (some small constant)
 * If the matrix is "fat" and "short", i.e. there are small number of rows and a
 * large number of columns,
 * there could be under-utilization of the hardware.
 * The template-ized version of this function is similar to what is found in
 * NVIDIA CUB
 * @param ReductionOp       Type of the functor object that implements the
 * reduction operation
 * @param AssignmentOp      Type of the functor object that is used to modify
 * the value before writing it to its final location in global memory for each
 * row
 */
template <typename ReductionOp, typename AssignmentOp, typename T>
__device__ void reduce_row(
    T *g_idata,  ///< input data stored in device memory (of size rows*cols)
    T *g_odata,  ///< output/temporary array store in device memory (of size
    /// rows*cols)
    unsigned int rows,  ///< rows in input and temporary/output arrays
    unsigned int cols,  ///< columns in input and temporary/output arrays
    ReductionOp
        reduction_op,  ///< Reduction operation to perform (functor object)
    AssignmentOp assignment_op,  ///< Operation to perform before assigning this
    /// to its final location in global memory for
    /// each row
    T initialValue) {  ///< initial value for the reduction variable
  // extern __shared__ T sdata[];
  extern __shared__ __align__(sizeof(T)) unsigned char my_sdata[];
  T *sdata = reinterpret_cast<T *>(my_sdata);

  // one block per row
  if (blockIdx.x >= rows) {
    return;
  }

  unsigned int block = blockIdx.x;
  unsigned int tid = threadIdx.x;
  unsigned int i = tid;
  unsigned int block_offset = block * cols;

  T v = initialValue;
  while (i < cols) {
    v = reduction_op(v, g_idata[block_offset + i]);
    i += blockDim.x;
  }

  // each thread puts its local sum into shared memory
  sdata[tid] = v;
  __syncthreads();

  // do reduction in shared mem
  if (blockDim.x >= 1024) {
    if (tid < 512) {
      sdata[tid] = v = reduction_op(v, sdata[tid + 512]);
    }
    __syncthreads();
  }
  if (blockDim.x >= 512) {
    if (tid < 256) {
      sdata[tid] = v = reduction_op(v, sdata[tid + 256]);
    }
    __syncthreads();
  }
  if (blockDim.x >= 256) {
    if (tid < 128) {
      sdata[tid] = v = reduction_op(v, sdata[tid + 128]);
    }
    __syncthreads();
  }
  if (blockDim.x >= 128) {
    if (tid < 64) {
      sdata[tid] = v = reduction_op(v, sdata[tid + 64]);
    }
    __syncthreads();
  }

  if (tid < 32) {
    // now that we are using warp-synchronous programming (below)
    // we need to declare our shared memory volatile so that the compiler
    // doesn't reorder stores to it and induce incorrect behavior.
    volatile T *smem = sdata;
    if (blockDim.x >= 64) {
      smem[tid] = v = reduction_op(v, smem[tid + 32]);
    }
    if (blockDim.x >= 32) {
      smem[tid] = v = reduction_op(v, smem[tid + 16]);
    }
    if (blockDim.x >= 16) {
      smem[tid] = v = reduction_op(v, smem[tid + 8]);
    }
    if (blockDim.x >= 8) {
      smem[tid] = v = reduction_op(v, smem[tid + 4]);
    }
    if (blockDim.x >= 4) {
      smem[tid] = v = reduction_op(v, smem[tid + 2]);
    }
    if (blockDim.x >= 2) {
      smem[tid] = v = reduction_op(v, smem[tid + 1]);
    }
  }

  // write result for this block to global mem, modify it with assignment op
  if (tid == 0) g_odata[block] = assignment_op(sdata[0]);
}

/**
 * Does a column wise reduction.
 * The intuition is that there are as many global threads as there are columns
 * Each global thread is responsible for a single element in the output vector
 * This of course leads to a under-utilization of the GPU resources.
 * For cases, where the number of columns is small, there can be unused SMs
 *
 * The template-ized version of this function is similar to what is found in
 * NVIDIA CUB
 * @param ReductionOp       Type of the functor object that implements the
 * reduction operation
 * @param AssignmentOp      Type of the functor object that is used to modify
 * the value before writing it to its final location in global memory for each
 * column
 */
template <typename ReductionOp, typename AssignmentOp, typename T>
__device__ void reduce_col(
    T *g_idata,  ///< input data stored in device memory (of size rows*cols)
    T *g_odata,  ///< output/temporary array store in device memory (of size
    /// rows*cols)
    unsigned int rows,  ///< rows in input and temporary/output arrays
    unsigned int cols,  ///< columns in input and temporary/output arrays
    ReductionOp
        reduction_op,  ///< Reduction operation to perform (functor object)
    AssignmentOp assignment_op,  ///< Operation to perform before assigning this
    /// to its final location in global memory for
    /// each column
    T initialValue)  ///< initial value for the reduction variable
{
  unsigned int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (global_tid >= cols) {
    return;
  }

  unsigned int i = global_tid;
  unsigned int grid_size = cols;
  T val = initialValue;

  while (i < rows * cols) {
    val = reduction_op(val, g_idata[i]);
    i += grid_size;
  }
  g_odata[global_tid] = assignment_op(val);
}

/**
 * Functor op for assignment op. This is a dummy/identity op.
 */
template <typename T>
struct IdentityOp {
  __device__ __forceinline__ T operator()(T a) const { return a; }
};

/**
 * Functor op for summation operation
 */
template <typename T>
struct SumOp {
  __device__ __forceinline__ T operator()(T a, T b) const { return a + b; }
};

/**
 * Do a summation over all elements of an array/matrix
 * @param g_idata   input data stored in device memory (of size n)
 * @param g_odata   output/temporary array stored in device memory (of size n)
 * @param n         size of the input and temporary/output arrays
 */
template <typename T>
__device__ void reduce_sum(T *g_idata, T *g_odata, unsigned int n) {
  SumOp<T> op;
  reduce<SumOp<T>, T>(g_idata, g_odata, n, op, (T)0.0);
}

extern "C" __global__ void reduce_sum_d(double *g_idata, double *g_odata,
                                        unsigned int n) {
  reduce_sum(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_sum_f(float *g_idata, float *g_odata,
                                        unsigned int n) {
  reduce_sum(g_idata, g_odata, n);
}

/**
 * Do a summation over all rows of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size rows)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template <typename T>
__device__ void reduce_row_sum(T *g_idata, T *g_odata, unsigned int rows,
                               unsigned int cols) {
  SumOp<T> op;
  IdentityOp<T> aop;
  reduce_row<SumOp<T>, IdentityOp<T>, T>(g_idata, g_odata, rows, cols, op, aop,
                                         0.0);
}

extern "C" __global__ void reduce_row_sum_d(double *g_idata, double *g_odata,
                                            unsigned int rows,
                                            unsigned int cols) {
  reduce_row_sum(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_row_sum_f(float *g_idata, float *g_odata,
                                            unsigned int rows,
                                            unsigned int cols) {
  reduce_row_sum(g_idata, g_odata, rows, cols);
}

/**
 * Do a summation over all columns of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size cols)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template <typename T>
__device__ void reduce_col_sum(T *g_idata, T *g_odata, unsigned int rows,
                               unsigned int cols) {
  SumOp<T> op;
  IdentityOp<T> aop;
  reduce_col<SumOp<T>, IdentityOp<T>, T>(g_idata, g_odata, rows, cols, op, aop,
                                         (T)0.0);
}

extern "C" __global__ void reduce_col_sum_d(double *g_idata, double *g_odata,
                                            unsigned int rows,
                                            unsigned int cols) {
  reduce_col_sum(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_col_sum_f(float *g_idata, float *g_odata,
                                            unsigned int rows,
                                            unsigned int cols) {
  reduce_col_sum(g_idata, g_odata, rows, cols);
}

/**
 * Functor op for max operation
 */
template <typename T>
struct MaxOp {
  __device__ __forceinline__ T operator()(T a, T b) const { return fmax(a, b); }
};

template <>
struct MaxOp<float> {
  __device__ __forceinline__ float operator()(float a, float b) const {
    return fmaxf(a, b);
  }
};

/**
 * Do a max over all elements of an array/matrix
 * @param g_idata   input data stored in device memory (of size n)
 * @param g_odata   output/temporary array stode in device memory (of size n)
 * @param n         size of the input and temporary/output arrays
 */
template <typename T>
__device__ void reduce_max(T *g_idata, T *g_odata, unsigned int n) {
  MaxOp<T> op;
  reduce<MaxOp<T>, T>(g_idata, g_odata, n, op, -MAX<T>());
}

extern "C" __global__ void reduce_max_d(double *g_idata, double *g_odata,
                                        unsigned int n) {
  reduce_max(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_max_f(float *g_idata, float *g_odata,
                                        unsigned int n) {
  reduce_max(g_idata, g_odata, n);
}

/**
 * Do a max over all rows of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size rows)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template <typename T>
__device__ void reduce_row_max(T *g_idata, T *g_odata, unsigned int rows,
                               unsigned int cols) {
  MaxOp<T> op;
  IdentityOp<T> aop;
  reduce_row<MaxOp<T>, IdentityOp<T>, T>(g_idata, g_odata, rows, cols, op, aop,
                                         -MAX<T>());
}

extern "C" __global__ void reduce_row_max_d(double *g_idata, double *g_odata,
                                            unsigned int rows,
                                            unsigned int cols) {
  reduce_row_max(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_row_max_f(float *g_idata, float *g_odata,
                                            unsigned int rows,
                                            unsigned int cols) {
  reduce_row_max(g_idata, g_odata, rows, cols);
}

/**
 * Do a max over all columns of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size cols)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template <typename T>
__device__ void reduce_col_max(T *g_idata, T *g_odata, unsigned int rows,
                               unsigned int cols) {
  MaxOp<T> op;
  IdentityOp<T> aop;
  reduce_col<MaxOp<T>, IdentityOp<T>, T>(g_idata, g_odata, rows, cols, op, aop,
                                         -MAX<T>());
}

extern "C" __global__ void reduce_col_max_d(double *g_idata, double *g_odata,
                                            unsigned int rows,
                                            unsigned int cols) {
  reduce_col_max(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_col_max_f(float *g_idata, float *g_odata,
                                            unsigned int rows,
                                            unsigned int cols) {
  reduce_col_max(g_idata, g_odata, rows, cols);
}

/**
 * Functor op for min operation
 */
template <typename T>
struct MinOp {
  __device__ __forceinline__ T operator()(T a, T b) const { return fmin(a, b); }
};

/**
 * Do a min over all elements of an array/matrix
 * @param g_idata   input data stored in device memory (of size n)
 * @param g_odata   output/temporary array stode in device memory (of size n)
 * @param n         size of the input and temporary/output arrays
 */
template <typename T>
__device__ void reduce_min(T *g_idata, T *g_odata, unsigned int n) {
  MinOp<T> op;
  reduce<MinOp<T>, T>(g_idata, g_odata, n, op, MAX<T>());
}

extern "C" __global__ void reduce_min_d(double *g_idata, double *g_odata,
                                        unsigned int n) {
  reduce_min(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_min_f(float *g_idata, float *g_odata,
                                        unsigned int n) {
  reduce_min(g_idata, g_odata, n);
}

/**
 * Do a min over all rows of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size rows)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template <typename T>
__device__ void reduce_row_min(T *g_idata, T *g_odata, unsigned int rows,
                               unsigned int cols) {
  MinOp<T> op;
  IdentityOp<T> aop;
  reduce_row<MinOp<T>, IdentityOp<T>, T>(g_idata, g_odata, rows, cols, op, aop,
                                         MAX<T>());
}

extern "C" __global__ void reduce_row_min_d(double *g_idata, double *g_odata,
                                            unsigned int rows,
                                            unsigned int cols) {
  reduce_row_min(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_row_min_f(float *g_idata, float *g_odata,
                                            unsigned int rows,
                                            unsigned int cols) {
  reduce_row_min(g_idata, g_odata, rows, cols);
}

/**
 * Do a min over all columns of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size cols)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template <typename T>
__device__ void reduce_col_min(T *g_idata, T *g_odata, unsigned int rows,
                               unsigned int cols) {
  MinOp<T> op;
  IdentityOp<T> aop;
  reduce_col<MinOp<T>, IdentityOp<T>, T>(g_idata, g_odata, rows, cols, op, aop,
                                         MAX<T>());
}

extern "C" __global__ void reduce_col_min_d(double *g_idata, double *g_odata,
                                            unsigned int rows,
                                            unsigned int cols) {
  reduce_col_min(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_col_min_f(float *g_idata, float *g_odata,
                                            unsigned int rows,
                                            unsigned int cols) {
  reduce_col_min(g_idata, g_odata, rows, cols);
}

/**
 * Functor op for product operation
 */
template <typename T>
struct ProductOp {
  __device__ __forceinline__ T operator()(T a, T b) const { return a * b; }
};

/**
 * Do a product over all elements of an array/matrix
 * @param g_idata   input data stored in device memory (of size n)
 * @param g_odata   output/temporary array stode in device memory (of size n)
 * @param n         size of the input and temporary/output arrays
 */
template <typename T>
__device__ void reduce_prod(T *g_idata, T *g_odata, unsigned int n) {
  ProductOp<T> op;
  reduce<ProductOp<T>, T>(g_idata, g_odata, n, op, (T)1.0);
}

extern "C" __global__ void reduce_prod_d(double *g_idata, double *g_odata,
                                         unsigned int n) {
  reduce_prod(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_prod_f(float *g_idata, float *g_odata,
                                         unsigned int n) {
  reduce_prod(g_idata, g_odata, n);
}

/**
 * Functor op for mean operation
 */
template <typename T>
struct MeanOp {
  const long
      _size;  ///< Number of elements by which to divide to calculate mean
  __device__ __forceinline__ MeanOp(long size) : _size(size) {}
  __device__ __forceinline__ T operator()(T total) const {
    return total / _size;
  }
};

/**
 * Do a mean over all rows of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size rows)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template <typename T>
__device__ void reduce_row_mean(T *g_idata, T *g_odata, unsigned int rows,
                                unsigned int cols) {
  SumOp<T> op;
  MeanOp<T> aop(cols);
  reduce_row<SumOp<T>, MeanOp<T>, T>(g_idata, g_odata, rows, cols, op, aop,
                                     (T)0.0);
}

extern "C" __global__ void reduce_row_mean_d(double *g_idata, double *g_odata,
                                             unsigned int rows,
                                             unsigned int cols) {
  reduce_row_mean(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_row_mean_f(float *g_idata, float *g_odata,
                                             unsigned int rows,
                                             unsigned int cols) {
  reduce_row_mean(g_idata, g_odata, rows, cols);
}

/**
 * Do a mean over all columns of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size cols)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template <typename T>
__device__ void reduce_col_mean(T *g_idata, T *g_odata, unsigned int rows,
                                unsigned int cols) {
  SumOp<T> op;
  MeanOp<T> aop(rows);
  reduce_col<SumOp<T>, MeanOp<T>, T>(g_idata, g_odata, rows, cols, op, aop,
                                     0.0);
}

extern "C" __global__ void reduce_col_mean_d(double *g_idata, double *g_odata,
                                             unsigned int rows,
                                             unsigned int cols) {
  reduce_col_mean(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_col_mean_f(float *g_idata, float *g_odata,
                                             unsigned int rows,
                                             unsigned int cols) {
  reduce_col_mean(g_idata, g_odata, rows, cols);
}

/**
 * Do an exp over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_exp(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = exp(A[index]);
  }
}

extern "C" __global__ void matrix_exp_d(double *A, double *C,
                                        unsigned int size) {
  matrix_exp(A, C, size);
}

extern "C" __global__ void matrix_exp_f(float *A, float *C, unsigned int size) {
  matrix_exp(A, C, size);
}

/**
 * Do an sqrt over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_sqrt(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = sqrt(A[index]);
  }
}

extern "C" __global__ void matrix_sqrt_d(double *A, double *C,
                                         unsigned int size) {
  matrix_sqrt(A, C, size);
}

extern "C" __global__ void matrix_sqrt_f(float *A, float *C,
                                         unsigned int size) {
  matrix_sqrt(A, C, size);
}

/**
 * Do an round over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_round(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = (T)llround(A[index]);
  }
}

extern "C" __global__ void matrix_round_d(double *A, double *C,
                                          unsigned int size) {
  matrix_round(A, C, size);
}

extern "C" __global__ void matrix_round_f(float *A, float *C,
                                          unsigned int size) {
  matrix_round(A, C, size);
}

/**
 * Do an abs over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_abs(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = (T)fabs(A[index]);
  }
}

extern "C" __global__ void matrix_abs_d(double *A, double *C,
                                        unsigned int size) {
  matrix_abs(A, C, size);
}

extern "C" __global__ void matrix_abs_f(float *A, float *C, unsigned int size) {
  matrix_abs(A, C, size);
}

/**
 * Do an log over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_log(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = log(A[index]);
  }
}

extern "C" __global__ void matrix_log_d(double *A, double *C,
                                        unsigned int size) {
  matrix_log(A, C, size);
}

extern "C" __global__ void matrix_log_f(float *A, float *C, unsigned int size) {
  matrix_log(A, C, size);
}

/**
 * Do an floor over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_floor(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = floor(A[index]);
  }
}

extern "C" __global__ void matrix_floor_d(double *A, double *C,
                                          unsigned int size) {
  matrix_floor(A, C, size);
}

extern "C" __global__ void matrix_floor_f(float *A, float *C,
                                          unsigned int size) {
  matrix_floor(A, C, size);
}

/**
 * Do an ceil over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_ceil(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = ceil(A[index]);
  }
}

extern "C" __global__ void matrix_ceil_d(double *A, double *C,
                                         unsigned int size) {
  matrix_ceil(A, C, size);
}

extern "C" __global__ void matrix_ceil_f(float *A, float *C,
                                         unsigned int size) {
  matrix_ceil(A, C, size);
}

/**
 * Do an sin over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_sin(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = sin(A[index]);
  }
}

extern "C" __global__ void matrix_sin_d(double *A, double *C,
                                        unsigned int size) {
  matrix_sin(A, C, size);
}

extern "C" __global__ void matrix_sin_f(float *A, float *C, unsigned int size) {
  matrix_sin(A, C, size);
}

/**
 * Do an sinh over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_sinh(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = sinh(A[index]);
  }
}

extern "C" __global__ void matrix_sinh_d(double *A, double *C,
                                         unsigned int size) {
  matrix_sinh(A, C, size);
}

extern "C" __global__ void matrix_sinh_f(float *A, float *C,
                                         unsigned int size) {
  matrix_sinh(A, C, size);
}

/**
 * Do an cos over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_cos(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = cos(A[index]);
  }
}

extern "C" __global__ void matrix_cos_d(double *A, double *C,
                                        unsigned int size) {
  matrix_cos(A, C, size);
}

extern "C" __global__ void matrix_cos_f(float *A, float *C, unsigned int size) {
  matrix_cos(A, C, size);
}

/**
 * Do an cosh over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_cosh(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = cosh(A[index]);
  }
}

extern "C" __global__ void matrix_cosh_d(double *A, double *C,
                                         unsigned int size) {
  matrix_cosh(A, C, size);
}

extern "C" __global__ void matrix_cosh_f(float *A, float *C,
                                         unsigned int size) {
  matrix_cosh(A, C, size);
}

/**
 * Do an tan over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_tan(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = tan(A[index]);
  }
}

extern "C" __global__ void matrix_tan_d(double *A, double *C,
                                        unsigned int size) {
  matrix_tan(A, C, size);
}

extern "C" __global__ void matrix_tan_f(float *A, float *C, unsigned int size) {
  matrix_tan(A, C, size);
}

/**
 * Do an tanh over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_tanh(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = tanh(A[index]);
  }
}

extern "C" __global__ void matrix_tanh_d(double *A, double *C,
                                         unsigned int size) {
  matrix_tanh(A, C, size);
}

extern "C" __global__ void matrix_tanh_f(float *A, float *C,
                                         unsigned int size) {
  matrix_tanh(A, C, size);
}

/**
 * Do an asin over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_asin(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = asin(A[index]);
  }
}

extern "C" __global__ void matrix_asin_d(double *A, double *C,
                                         unsigned int size) {
  matrix_asin(A, C, size);
}

extern "C" __global__ void matrix_asin_f(float *A, float *C,
                                         unsigned int size) {
  matrix_asin(A, C, size);
}

/**
 * Do an acos over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_acos(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = acos(A[index]);
  }
}

extern "C" __global__ void matrix_acos_d(double *A, double *C,
                                         unsigned int size) {
  matrix_acos(A, C, size);
}

extern "C" __global__ void matrix_acos_f(float *A, float *C,
                                         unsigned int size) {
  matrix_acos(A, C, size);
}

/**
 * Do an atan over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_atan(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = atan(A[index]);
  }
}

extern "C" __global__ void matrix_atan_d(double *A, double *C,
                                         unsigned int size) {
  matrix_atan(A, C, size);
}

extern "C" __global__ void matrix_atan_f(float *A, float *C,
                                         unsigned int size) {
  matrix_atan(A, C, size);
}

/**
 * Do an sign over all the elements of a matrix
 * Assign -1, 0 or 1 depending on the element being negative, 0 or positive
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_sign(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    if (A[index] == 0.0) {
      C[index] = 0.0;
    } else {
      C[index] = copysign(1.0, A[index]);
    }
  }
}

extern "C" __global__ void matrix_sign_d(double *A, double *C,
                                         unsigned int size) {
  matrix_sign(A, C, size);
}

extern "C" __global__ void matrix_sign_f(float *A, float *C,
                                         unsigned int size) {
  matrix_sign(A, C, size);
}

/**
 * Do an sigmoid over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template <typename T>
__device__ void matrix_sigmoid(T *A, T *C, unsigned int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    C[index] = 0.5 * tanh(0.5 * A[index]) + 0.5;
  }
}

extern "C" __global__ void matrix_sigmoid_d(double *A, double *C,
                                         unsigned int size) {
  matrix_sigmoid(A, C, size);
}

extern "C" __global__ void matrix_sigmoid_f(float *A, float *C,
                                         unsigned int size) {
  matrix_sigmoid(A, C, size);
}
